// %%cuda --name snake-super.cu
/*	CS6023 GPU Programming
 	Project - Genetic Algorithm to optimise snakes game
 		Done By, 
 		Shivam Mittal, cs16b038
        Rachit Tibrewal, cs16b022
        R Sai Harshini, cs16b112
 	Parallel Code
*/


#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

using namespace std;

#define cudaErrorTrace() {\
    hipError_t err = hipGetLastError();\
    if(err != hipSuccess) {\
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);\
        exit(EXIT_FAILURE);\
    }\
}

typedef pair<int, int> ii;

#define POPULATION_SIZE 4096
#define NUM_GENERATIONS 300
#define NUM_FOODS 256

// Defining the parameters in each layer of the neural network.
// int n = 24, m1 = 16, o = 4;
#define n 24
#define m1 16
#define o 4

std::random_device oracle{};
auto tempo = oracle();    
mt19937 rd{tempo};

int GENOME_LENGTH;

float *organism;

int *fitness_score = NULL, max_score;

#define M 80
#define N 80
#define Q_LEN 128

__device__
bool check(int u, int v, int i, int j) {
	if(u == 0 && v != 0) {
		if(i == u && j == v / abs(v)) {
			return true;
		}
	}
	else if(u != 0 && v == 0) {
		if(i == u / abs(u) && j == v) {
			return true;
		}
	}
	else if(u != 0 && v != 0) {
		if(i == u / abs(u) && j == v / abs(v)) {
			return true;
		}
	}
	return false;
}

__global__
void evaluate(float* genes, int* foods, int* fitness_score, int GENOME_LENGTH) {
	int food_pos[2];
	int snake[Q_LEN][2];
	int snake_init_length;
	int st;
	int en;
	int maxiters;
	int additers;
	int loops;
	int snake_motion;
	int dir[2];
	int score;
	int fi;
	__shared__ float dist[8][3];
	float* input;
	float* W1;
	float* b1;
	float* W2;
	float* b2;
	__shared__ float output1[m1];
	__shared__ float output2[o];
	__shared__ int com;
	int head[2];
	int snake_size;
	int x, y;
	int snakeIsAlive;
	int foodEaten;
	
	extern __shared__ float gene[];

	int i = threadIdx.x;

	while(i < GENOME_LENGTH) {
		gene[i] = genes[blockIdx.x * GENOME_LENGTH + i];
		i += blockDim.x;
	}
	__syncthreads();
	int init_x = M / 2;
	int init_y = N / 2;

	input = &dist[0][0];
	W1 = &gene[0];
	b1 = &gene[n * m1];
	W2 = &gene[n * m1 + m1];
	b2 = &gene[n * m1 + m1 + m1 * o];
	snake_init_length = 5;
	st = 0;
	en = snake_init_length;
	maxiters = 3 * (M + N);
	additers = 1 * (M + N);
	loops = maxiters;
	score = 0;
	fi = 0;
	snake_motion = 3;
	dir[0] = 1;
	dir[1] = 0;
	snakeIsAlive = 1;
	foodEaten = 1;

	__syncthreads();
	for(int i = 0; i < snake_init_length; i++) {
		snake[i][0] = i + init_x;
		snake[i][1] = init_y;
	}
	__syncthreads();
	do
	{
		if(foodEaten) {
			food_pos[0] = foods[2 * fi];
			food_pos[1] = foods[2 * fi + 1];
			fi++;
			foodEaten = 0; 
		}

		head[0] = snake[(en - 1 + Q_LEN) % Q_LEN][0];
		head[1] = snake[(en - 1 + Q_LEN) % Q_LEN][1]; 
		x = head[0];
		y = head[1];
		snake_size = (en - st + Q_LEN) % Q_LEN;
		for(int i = 0; i < 8; i++) {
			for(int j = 0; j < 3; j++) {
				dist[i][j] = 2 * max(M, N);
			}
		}
		__syncthreads();
		if (threadIdx.x < 9 && threadIdx.x != 4) {
			int i = threadIdx.x / 3 - 1;
			int j = threadIdx.x % 3 - 1;
			int k = (threadIdx.x > 4) ? (threadIdx.x - 1) : threadIdx.x;
			if(i == 0) {
				dist[k][0] = (j > 0) * N - j * y;
			} else if(j == 0) {
				dist[k][0] = (i > 0) * M - i * x;
			} else {
				dist[k][0] = min((i > 0) * M - i * x, (j > 0) * N - j * y);
			}

			int u, v;
			u = food_pos[0] - x;
			v = food_pos[1] - y;
			if(check(u, v, i, j)) {
				if(abs(dist[k][1]) > float(abs(u) + abs(v)) / (abs(i) + abs(j))) {
					dist[k][1] = float(abs(u) + abs(v)) / (abs(i) + abs(j));
				}
			}

			for(int ti = 0; ti < snake_size; ti++) {
				int haha[2];
				haha[0] = snake[st][0];
				haha[1] = snake[st][1];
				// snake.pop();
				st = (st + 1 + Q_LEN) % Q_LEN;
				u = haha[0] - x;
				v = haha[1] - y;
				if(check(u, v, i, j)) {
					if(abs(dist[k][2]) > float(abs(u) + abs(v))/(abs(i) + abs(j))) {
						dist[k][2] = float(abs(u) + abs(v))/(abs(i) + abs(j));
					}
				}
				snake[en][0] = haha[0];
				snake[en][1] = haha[1];
				en = (en + 1 + Q_LEN) % Q_LEN;
				// snake.push(haha);
			}
		}
		__syncthreads();
		if(threadIdx.x < m1) {
			int i = threadIdx.x;
			/* dense 1 */
			output1[i] = 0;
			for(int j = 0; j < n; j++) {
				output1[i] += W1[j * m1 + i] * input[j];
			}
			output1[i] += b1[i];
			/* sigmoid */
		
			output1[i] = 1.0 / (1.0 + expf(-output1[i]));
		}
		__syncthreads();
		if (threadIdx.x < o) {
			int i = threadIdx.x;
			/* dense 2 */
			output2[i] = 0;
			for(int j = 0; j < m1; j++) {
				output2[i] += W2[j * o + i] * output1[j];
			}
			output2[i] += b2[i];

			/* sigmoid */
			output2[i] = 1.0 / (1.0 + expf(-output2[i]));
		}
		__syncthreads();
		if(threadIdx.x == 0) {
			float maxm = output2[0];
			com = 0;
			for(int i = 1; i < o; i++) {
				if (output2[i] > maxm) {
					maxm = output2[i];
					com = i;
				}
			}	
		}
		__syncthreads();
		if(com == 0) {
			// no change to direction
		} else if(com == 1) {
			if(snake_motion == 1) {
				// change to west
				snake_motion = 4;
				dir[0] = -1;
				dir[1] = 0;
			} else if(snake_motion == 2) {
				// change to east
				snake_motion = 3;
				// dir = ii(1, 1.0f);
				dir[0] = 1;
				dir[1] = 0;
			} else if(snake_motion == 3) {
				// change to north
				snake_motion = 1;
				// dir = ii(0, -1);
				dir[0] = 0;
				dir[1] = -1;
			} else if(snake_motion == 4) {
				// change to south
				snake_motion = 2;
				// dir = ii(0, 1);
				dir[0] = 0;
				dir[1] = 1;
			}
		} else if(com == 2){
			if(snake_motion == 1) {
				// change to east
				snake_motion = 3;
				// dir = ii(1, 0);
				dir[0] = 1;
				dir[1] = 0;
			} else if(snake_motion == 2) {
				// change to west
				snake_motion = 4;
				// dir = ii(-1, 0);
				dir[0] = -1;
				dir[1] = 0;
			} else if(snake_motion == 3) {
				// change to south
				snake_motion = 2;
				// dir = ii(0, 1);
				dir[0] = 0;
				dir[1] = 1;
			} else if(snake_motion == 4) {
				// change to north
				snake_motion = 1;
				// dir = ii(0, -1);
				dir[0] = 0;
				dir[1] = -1;
			}
		} else if(com == 3) {
			snakeIsAlive = 0;
			// break;
		}
		
		// check if the snake eats the food in the next move
		// head = ii(head.first + dir.first, head.second + dir.second); 
		head[0] = head[0] + dir[0];
		head[1] = head[1] + dir[1];
		snake[en][0] = head[0];
		snake[en][1] = head[1];
		en = (en + 1 + Q_LEN) % Q_LEN;

		// move the snake in the direction
		if(head[0] != food_pos[0] || head[1] != food_pos[1]) {
			st = (st + 1 + Q_LEN) % Q_LEN;
		} else {
			score += 1;
			loops += additers;
			foodEaten = 1;
		}

		// check if the snake crosses any boundaries
		x = head[0];
		y = head[1];
		if(x < 0 || y < 0 || x >= M || y >= N) {
			// crossed the boundart game over
			snakeIsAlive = 0;
			// break;
		}

		// check if the snake eats it self
		snake_size = (en - st + Q_LEN) % Q_LEN;
		for(int i = 0; i < snake_size; i++) {
			int haha[2];
			haha[0] = snake[st][0];
			haha[1] = snake[st][1];
			// snake.pop();
			st = (st + 1 + Q_LEN) % Q_LEN;
			if(i < snake_size - 1 && haha[0] == x && haha[1] == y) {
				snakeIsAlive = 0;
				break;
			}    
			snake[en][0] = haha[0];
			snake[en][1] = haha[1];
			en = (en + 1 + Q_LEN) % Q_LEN;        
			// snake.push(haha);
		}			
		loops--;
	} while(snakeIsAlive && loops >= 0 && fi < NUM_FOODS);
	
	__syncthreads();

	if(threadIdx.x == 0) {
		fitness_score[blockIdx.x] = score;
	}
}

// Function to select the best (selection_cutoff)% of the population in each generation where the organisms are sorted in the decreasing of the fitness scores.
int selection(float selection_cutoff) {
	int selected = 0;
	float *new_generation = (float *) malloc(sizeof(float) * POPULATION_SIZE * GENOME_LENGTH);
	
	ii list[POPULATION_SIZE];
	
	for(int i = 0; i < POPULATION_SIZE; i++) {
		list[i] = {-fitness_score[i], i};
	}

	sort(list, list + POPULATION_SIZE);

	for(int i = 0; i < int(POPULATION_SIZE * selection_cutoff); i++) {
		copy(organism + list[i].second * GENOME_LENGTH, organism + (list[i].second + 1) * GENOME_LENGTH, new_generation + selected * GENOME_LENGTH);
		selected++;
	}

	free(organism);
	organism = new_generation;
	return selected;
}


// Function to crossover between the the best population selected by the selection function and create the next generation.
// The next generation comprises of the best population selected in the current generation and the organisms generated by their crossover.
__global__
void crossover(unsigned int *rand1, unsigned int *rand2, float *d_organism, const int offset) {
	int idx = blockIdx.x;
	
	if(threadIdx.x <= rand2[idx] % blockDim.x) {
		d_organism[(offset + blockIdx.x) * blockDim.x + threadIdx.x] = d_organism[(rand1[idx] % offset) * blockDim.x + threadIdx.x];
	} else {
		d_organism[(offset + blockIdx.x) * blockDim.x + threadIdx.x] = d_organism[(rand1[blockDim.x + idx] % offset) * blockDim.x + threadIdx.x];
	}
}

// Function to mutate the genomes of each organism. Mutation is one of the fundamental concept of genetic algorithms. 
__global__
void mutate(float *rand1, float *rand2, float *d_organism, const float mutation_rate) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	
	float changed = d_organism[idx];

	if(rand1[idx] < mutation_rate) {
		changed += rand2[idx] / 5.0;
 		changed = max(-1.0f, changed);
 		changed = min(1.0f, changed);
	}

	d_organism[idx] = changed;
}

__global__ 
void scale(float* mat, float a, float b) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	mat[idx] = (mat[idx] - 0.5) * (b - a) + (b + a) / 2.0;
}

int main() {
	srand(time(NULL));

    GENOME_LENGTH = n * m1 + m1 + m1 * o + o;
	
	printf("Genome length: %d\n", GENOME_LENGTH);
	printf("Generation size: %d\n", POPULATION_SIZE);

	const int L = POPULATION_SIZE * GENOME_LENGTH;
	const size_t data_size = sizeof(float) * POPULATION_SIZE * GENOME_LENGTH;
	
	organism = (float *) malloc(data_size);
	
	int blocks = 4096;
    int threads = GENOME_LENGTH;
	
	float* d_organism;

	/* Allocate memory for organisms on device */
	hipMalloc((void**) &d_organism, data_size);
	cudaErrorTrace();

	hiprandGenerator_t prng;
	
	/* Create pseudo random number generator */
	hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_MT19937);
	cudaErrorTrace();

	/* Set seed */
	hiprandSetPseudoRandomGeneratorSeed(prng, 42ULL);
	cudaErrorTrace();
	
	/** 
	* Create genomes by uniform initialization of organism matrix
	*/
	hiprandGenerateUniform(prng, d_organism, L);
	cudaErrorTrace();	
	
	/* adjust the range of uniform value to (-1, 1] */
	scale<<<blocks, threads>>>(d_organism, -1.0, 1.0);

	/* Copy device memory to host */
	hipMemcpy(organism, d_organism, data_size, hipMemcpyDeviceToHost);
	cudaErrorTrace();

	max_score = 0;

	FILE *fout = fopen("genomes.txt", "w");

	fprintf(fout, "NUM_GENERATIONS = %d\n", NUM_GENERATIONS);
	fprintf(fout, "POPULATION_SIZE = %d\n", POPULATION_SIZE);
	fprintf(fout, "GENOME_LENGTH = %d\n", GENOME_LENGTH);

	int* d_fitness_score;

	/* Allocate memory for fitness score on host */
	fitness_score = (int*) malloc(sizeof(int) * POPULATION_SIZE);

	/* Allocate memory for fitness score on device */
	hipMalloc((void**) &d_fitness_score, sizeof(int) * POPULATION_SIZE);
	cudaErrorTrace();
	
	const size_t food_size = sizeof(int) * 2 * NUM_FOODS;
	
	int *h_foods, *d_foods;
	
	h_foods = (int*) malloc(food_size);
	hipMalloc((void**) &d_foods, food_size);
	cudaErrorTrace();
	
	unsigned int *random_uints[2];
	float *random_floats[2];
	
	hipMalloc((void**) &random_uints[0], sizeof(int) * 2 * POPULATION_SIZE);
	hipMalloc((void**) &random_uints[1], sizeof(int) * POPULATION_SIZE);
	hipMalloc((void**) &random_floats[0], sizeof(int) * POPULATION_SIZE * GENOME_LENGTH);
	hipMalloc((void**) &random_floats[1], sizeof(int) * POPULATION_SIZE * GENOME_LENGTH);

	for(int i = 0; i < NUM_GENERATIONS; i++) {
		int local_max = -1, local_best;

		for(int k = 0; k < NUM_FOODS; k++) {
			h_foods[2 * k] = rand() % M;
			h_foods[2 * k + 1] = rand() % N;
		}

		/* Copy food positions from host to device */
		hipMemcpy(d_foods, h_foods, food_size, hipMemcpyHostToDevice);
		cudaErrorTrace();
		
		/* Copy host organism to device */
		hipMemcpy(d_organism, organism, data_size, hipMemcpyHostToDevice);
		
		/**
		Logic:
		Each organism is allocated a block,
		the threads in the blocks perform
		the operations for the organism 
		*/
		blocks = POPULATION_SIZE;
		threads = 32;

		evaluate<<<blocks, threads, sizeof(float) * GENOME_LENGTH>>>(d_organism, d_foods, d_fitness_score, GENOME_LENGTH);

		hipMemcpy(fitness_score, d_fitness_score, sizeof(int) * POPULATION_SIZE, hipMemcpyDeviceToHost);
		cudaErrorTrace();
		
		for(int j = 0; j < POPULATION_SIZE; j++) {
			if(local_max < fitness_score[j]) {
				local_max = fitness_score[j];
				local_best = j;
			}
		}

		// printing the genome of the best organism in the generation to the file.
		for(int k = 0; k < GENOME_LENGTH; k++) {
			fprintf(fout, "%f ", organism[local_best * GENOME_LENGTH + k]);
		}
		fprintf(fout, "\n");

		max_score = max(max_score, local_max);

		printf("Score after generation %d => local: %d | max: %d\n", i, local_max, max_score);

		int ga_blocks = POPULATION_SIZE;
		int ga_threads = GENOME_LENGTH;

		int selected = selection(0.15);

		hipMemcpy(d_organism, organism, data_size, hipMemcpyHostToDevice);		
		
		hiprandGenerate(prng, random_uints[0], 2 * POPULATION_SIZE);
		hiprandGenerate(prng, random_uints[1], POPULATION_SIZE);		
		
		crossover<<<POPULATION_SIZE - selected, GENOME_LENGTH>>>(random_uints[0], random_uints[1], d_organism, selected);

		hiprandGenerateUniform(prng, random_floats[0], POPULATION_SIZE * GENOME_LENGTH);
		hiprandGenerateNormal(prng, random_floats[1], POPULATION_SIZE * GENOME_LENGTH, 0.0, 1.0);
		
		mutate<<<ga_blocks, ga_threads>>>(random_floats[0], random_floats[1], d_organism, 1e-2);

		hipMemcpy(organism, d_organism, data_size, hipMemcpyDeviceToHost);
	}

	hipFree(d_organism);
	hipFree(d_fitness_score);
	hipFree(d_foods);
	hipFree(random_uints[0]);
	hipFree(random_uints[1]);
	hipFree(random_floats[0]);
	hipFree(random_floats[1]);

	free(organism);

	return 0;
}